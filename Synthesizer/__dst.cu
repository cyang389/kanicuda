#include "hip/hip_runtime.h"
# 1 "__cp.cu"
# 1 "<built-in>" 1
# 1 "<built-in>" 3
# 360 "<built-in>" 3
# 1 "<command line>" 1
# 1 "<built-in>" 2
# 1 "__cp.cu" 2







__global__ void diffusion_kernel(float* in,
                                 float* out,
                                 int nx, int ny, int nz,
                                 float ce, float cw, float cn, float cs,
                                 float ct, float cb, float cc) {
  profile("threadIdx.x threadIdx.y blockDim.x blockDim.y csb c i j");
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  __shared__ float sb[3 * 4];
  int csb = threadIdx.x + threadIdx.y * blockDim.x;
  for (int k = 0; k < nz; ++k) {
    sb[csb] = in[c];
    int w = (i == 0)        ? c : c - 1;
    int e = (i == nx-1)     ? c : c + 1;
    int n = (j == 0)        ? c : c - nx;
    int s = (j == ny-1)     ? c : c + nx;
    int b = (k == 0)        ? c : c - xy;
    int t = (k == nz-1)     ? c : c + xy;
    out[c] = 
        cc * in[c] 
      + cw * __opt__951353.in[w] 
      + ce * __opt__217004.in[e] 
      + cs * __opt__968837.in[s]
      + cn * __opt__636769.in[n] 
      + cb * in[b] 
      + ct * in[t];   
    c += xy;
  }
}

void initialize(float *buff, const int nx, const int ny, const int nz,
                const float kx, const float ky, const float kz,
                const float dx, const float dy, const float dz,
                const float kappa, const float time) {
  float ax = exp(-kappa*time*(kx*kx));
  float ay = exp(-kappa*time*(ky*ky));
  float az = exp(-kappa*time*(kz*kz));
  int jz;  
  for (jz = 0; jz < nz; jz++) {
    int jy;
    for (jy = 0; jy < ny; jy++) {
      int jx;
      for (jx = 0; jx < nx; jx++) {
        int j = jz*nx*ny + jy*nx + jx;
        float x = dx*((float)(jx + 0.5));
        float y = dy*((float)(jy + 0.5));
        float z = dz*((float)(jz + 0.5));
        float f0 = (float)0.125
          *(1.0 - ax*cos(kx*x))
          *(1.0 - ay*cos(ky*y))
          *(1.0 - az*cos(kz*z));
        buff[j] = __symbol();//f0;
      }
    }
  }
}


int main(){
  int count = 3;
  int nx, ny, nz;
  nx = 3*3;
  ny = 4*3;
  nz = 4;

  float l, kappa;
  float kx, ky, kz;
  float dx, dy, dz, dt;
  float ce, cw, cn, cs, ct, cb, cc;
  l = 1.0;
  kappa = 0.1;
  dx = l / nx;
  dy = l / ny;
  dz = l / nz;
  kx = 2.0 * (3.1415926535897932384626);
  ky = 2.0 * (3.1415926535897932384626);
  kz = 2.0 * (3.1415926535897932384626);
  dt = 0.1 * dx * dy / kappa;
  ce = kappa * dt /(dx*dx);
  cw = kappa * dt /(dx*dx);
  cn = kappa * dt /(dy*dy);
  cs = kappa * dt /(dy*dy);
  ct = kappa * dt /(dz*dz);
  cb = kappa * dt /(dz*dz);
  cc = 1.0 - (ce+cw+cn+cs+ct+cb);

  float *in, *dev_in, *dev_out;
  int s = sizeof(float) * nx * ny * nz;
  in = (float *)malloc(s);
  initialize(in, nx, ny, nz,
             kx, ky, kz, dx, dy, dz,
             kappa, 0.0);
  hipMalloc((void**)&dev_in, s);
  hipMalloc((void**)&dev_out, s);
  hipMemcpy(dev_in, in, s, hipMemcpyHostToDevice);


  dim3 block(3, 4, 1);
  dim3 grid(3, 3, 1);

  
# 120 "__cp.cu"


  //Stopwatch st;
  //StopwatchStart(&st);

  for(int i = 0; i < count; i++){
    diffusion_kernel<<<grid, block>>>(
      dev_in, dev_out, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
    float *t;
    t = dev_in;
    dev_in = dev_out;
    dev_out = t;
  }
  hipMemcpy(in, dev_in, s, hipMemcpyDeviceToHost);

  //float elapsed_time = StopwatchStop(&st);

  printf("pass\n");
  //printf("kernel time: %f\n", elapsed_time);
  
# 149 "__cp.cu"

  
  return(0);
}

